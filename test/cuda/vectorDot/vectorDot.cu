
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void dot_product_kernel(const double* A, const double* B, double* partial_sum, int N) {
    __shared__ double cache[256];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIdx = threadIdx.x;

    double temp = 0.0;
    while (tid < N) {
        temp += A[tid] * B[tid];
        tid += blockDim.x * gridDim.x;
    }

    cache[cacheIdx] = temp;
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (cacheIdx < stride) {
            cache[cacheIdx] += cache[cacheIdx + stride];
        }
        __syncthreads();
    }

    if (cacheIdx == 0)
        partial_sum[blockIdx.x] = cache[0];
}

int main() {
    const int N = 1 << 20;  
    const int BLOCKS = 128;
    const int THREADS = 256;
    const int trials = 10;

    size_t size = N * sizeof(double);
    double *h_A = (double*)malloc(size);
    double *h_B = (double*)malloc(size);

    for (int i = 0; i < N; ++i) {
        h_A[i] = 1.0;
        h_B[i] = 2.0;
    }

    double *d_A, *d_B, *d_partial;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_partial, BLOCKS * sizeof(double));

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    double total_time = 0.0;

    for (int i = 0; i < trials; ++i) {
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);

        dot_product_kernel<<<BLOCKS, THREADS>>>(d_A, d_B, d_partial, N);

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float ms = 0;
        hipEventElapsedTime(&ms, start, stop);
        total_time += ms / 1000.0;

        double h_partial[BLOCKS];
        hipMemcpy(h_partial, d_partial, BLOCKS * sizeof(double), hipMemcpyDeviceToHost);
        double result = 0.0;
        for (int j = 0; j < BLOCKS; ++j)
            result += h_partial[j];

        // printf("Trial %d: Time = %.6f s, Result = %.2f\n", i + 1, ms / 1000.0, result);

        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    printf("Average Time: %.6f s\n", total_time / trials);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_partial);
    free(h_A);
    free(h_B);

    return 0;
}
