#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void dsum_kernel(const double* A, double* partial_sum, int N) {
    __shared__ double cache[256];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIdx = threadIdx.x;

    double temp = 0.0;
    while (tid < N) {
        temp += A[tid];
        tid += blockDim.x * gridDim.x;
    }

    cache[cacheIdx] = temp;
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (cacheIdx < stride) {
            cache[cacheIdx] += cache[cacheIdx + stride];
        }
        __syncthreads();
    }

    if (cacheIdx == 0)
        partial_sum[blockIdx.x] = cache[0];
}

int main() {
    const int N = 1 << 20;     // 1048576 个 double
    const int BLOCKS = 128;
    const int THREADS = 256;
    const int trials = 10;

    size_t size = N * sizeof(double);
    double *h_A = (double*)malloc(size);

    for (int i = 0; i < N; ++i) {
        h_A[i] = (double)rand() / RAND_MAX;  
    }

    double *d_A, *d_partial;
    hipMalloc(&d_A, size);
    hipMalloc(&d_partial, BLOCKS * sizeof(double));

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    double total_time = 0.0;

    for (int i = 0; i < trials; ++i) {
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);


        dsum_kernel<<<BLOCKS, THREADS>>>(d_A, d_partial, N);

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float ms = 0;
        hipEventElapsedTime(&ms, start, stop);
        total_time += ms / 1000.0;

        double h_partial[BLOCKS];
        hipMemcpy(h_partial, d_partial, BLOCKS * sizeof(double), hipMemcpyDeviceToHost);

        double sum = 0.0;
        for (int j = 0; j < BLOCKS; ++j)
            sum += h_partial[j];

        printf("Trial %d: sum = %.0f\n", i + 1, sum);

        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    printf("Average Time: %.6f s\n", total_time / trials);

    hipFree(d_A);
    hipFree(d_partial);
    free(h_A);

    return 0;
}
