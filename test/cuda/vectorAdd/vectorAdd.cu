
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vector_add_kernel(const double* A, const double* B, double* C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    const int N = 1 << 20;  // 1M
    const int trials = 10;
    const int threadsPerBlock = 256;
    const int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    size_t size = N * sizeof(double);
    double *h_A = (double*)malloc(size);
    double *h_B = (double*)malloc(size);
    double *h_C = (double*)malloc(size);

    for (int i = 0; i < N; ++i) {
        h_A[i] = 1.0;
        h_B[i] = 2.0;
    }

    double *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    float total_time = 0.0;

    for (int i = 0; i < trials; ++i) {
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);

        vector_add_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float ms;
        hipEventElapsedTime(&ms, start, stop);
        total_time += ms / 1000.0;

        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // printf("Sample Result C[0] = %.2f\n", h_C[0]);
    printf("Average Time: %.6f s\n", total_time / trials);

    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}
